#include "hip/hip_runtime.h"
// Copyright (c) 2015, Sebastien Sydney Robert Bigot
// All rights reserved.
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <vector>

#include <hip/hip_complex.h>

#include <cstdio>

#define SYNC(EXPR) do {EXPR; hipDeviceSynchronize();} while(false)
//#define SYNC(EXPR) EXPR

typedef int32_t color_t;

namespace {
  
  extern __shared__ float shmem[];

  template <typename T>
  class KernelArray
  {  
  public:
    KernelArray(thrust::device_vector<T> dVec)
     :  _array(thrust::raw_pointer_cast(dVec.data())),
        _size(dVec.size())
    {}
        
    __device__
    T &operator[](int i)
    {
      return _array[i];
    }

    __device__
    const T &operator[](int i) const
    {
      return _array[i];
    }

    __host__ __device__
    size_t size() const
    {
      return _size;
    }

  private:
    T* const _array;
    const size_t _size;
  };
   

  // This kernel gather the ith (i=0,1,2) vertices x and y coords for all triangles:
  __global__
  void GatherVertexCoordinates( 
    const KernelArray<int> triangles,                                                               
    const KernelArray<float> vertexCoords,
    KernelArray<float> gatheredxs,
    KernelArray<float> gatheredys,
    int vi
  ) {
    const int myTriangleId = blockIdx.x * blockDim.x + threadIdx.x;
    const int nbTriangle = triangles.size() / 3;
    if (myTriangleId < nbTriangle) {
      // Lookup the global index of the ith vertex of my triangle 
      int vertexId = triangles[3 * myTriangleId + vi];
      // Lookup the coord in the vertex coords array
      gatheredxs[myTriangleId] = vertexCoords[2 * vertexId];
      gatheredys[myTriangleId] = vertexCoords[2 * vertexId + 1];
    }
  }

  __global__
  void Assemble(
    const KernelArray<float> coeffs, 
    const KernelArray<color_t> colors, 
    const KernelArray<int> tt, 
    KernelArray<float> result,
    color_t color) {
    const int myTriangleId = blockIdx.x * blockDim.x + threadIdx.x;
    if (myTriangleId < colors.size() && colors[myTriangleId] == color) {
      const int *mytt = &tt[9 * myTriangleId];
      const float *myCoeffs = &coeffs[9 * myTriangleId];
      result[mytt[0]] += myCoeffs[0];
      result[mytt[1]] += myCoeffs[1];
      result[mytt[2]] += myCoeffs[2];
      result[mytt[3]] += myCoeffs[3];
      result[mytt[4]] += myCoeffs[4];
      result[mytt[5]] += myCoeffs[5];
      result[mytt[6]] += myCoeffs[6];
      result[mytt[7]] += myCoeffs[7];
      result[mytt[8]] += myCoeffs[8];
    }
  }

  __global__
  void ComputeElementsCoeffs( 
    const KernelArray<float> q1xs,
    const KernelArray<float> q1ys, 
    const KernelArray<float> q2xs,
    const KernelArray<float> q2ys,
    const KernelArray<float> q3xs, 
    const KernelArray<float> q3ys,
    KernelArray<float> coefficients
  ) {
 
    const size_t nbTriangles = q1xs.size();
    const int myTriangleId = blockIdx.x * blockDim.x + threadIdx.x;
    float *shcoeffs = &shmem[9 * threadIdx.x];
    if (myTriangleId < nbTriangles) {

      float ux = q2xs[myTriangleId] - q3xs[myTriangleId];
      float uy = q2ys[myTriangleId] - q3ys[myTriangleId];
      float vx = q3xs[myTriangleId] - q1xs[myTriangleId];
      float vy = q3ys[myTriangleId] - q1ys[myTriangleId];
      float wx = q1xs[myTriangleId] - q2xs[myTriangleId];
      float wy = q1ys[myTriangleId] - q2ys[myTriangleId];

      float area = 0.5f * (ux * vy - uy * vx);
      float prefactor = -1 /  (4 * area);

      float uu = prefactor * (ux * ux + uy * uy);
      float uv = prefactor * (ux * vx + uy * vy);
      float uw = prefactor * (ux * wx + uy * wy);
      float vv = prefactor * (vx * vx + vy * vy);
      float vw = prefactor * (vx * wx + vy * wy);
      float ww = prefactor * (wx * wx + wy * wy);

      float mDiag = area / 6, m = mDiag / 2;

      shcoeffs[0] = uu + mDiag;
      shcoeffs[1] = uv + m;
      shcoeffs[2] = uw + m;
      shcoeffs[3] = uv + m;
      shcoeffs[4] = vv + mDiag;
      shcoeffs[5] = vw + m;
      shcoeffs[6] = uw + m;
      shcoeffs[7] = vw + m;
      shcoeffs[8] = ww + mDiag;
    }

    __syncthreads();

    // Coalesced write
    int coeffsPerBlock = 9 * blockDim.x, totalCoeff = 9 * nbTriangles;
    for (int local = threadIdx.x, global = blockIdx.x * coeffsPerBlock + threadIdx.x;
          local < coeffsPerBlock && global < totalCoeff;
          global += blockDim.x, local += blockDim.x) {
          coefficients[global] = shmem[local];
    }

  }

}

extern "C"
{
  void do_global_assembly_on_gpu(
    size_t nv,
    size_t nt,
    const double *vs,
    const int32_t *ts,
    const color_t *colors,
    const int32_t *rowptr,
    const int32_t *colidx,
    const int32_t *tt,
    double *coeffs
  ) 
  { 
    using namespace thrust;

    // Push triangles and vertices on the GPU
    device_vector<float> vertexCoordsOnGpu(vs, &vs[2 * nv]);
    device_vector<int> triangleVidsOnGpu(ts, &ts[3 * nt]);
  
    int trianglesPerBlock = 512,  nbBlock = 1 + nt / trianglesPerBlock;

    // Pack triangle coordinates xs, ys for each edges u, v, w
    device_vector<float> q1xs(nt), q1ys(nt), q2xs(nt), q2ys(nt), q3xs(nt), q3ys(nt);
    SYNC((
      GatherVertexCoordinates<<<nbBlock, trianglesPerBlock>>>(
        triangleVidsOnGpu,                                                                                                                         
        vertexCoordsOnGpu,
        q1xs,
        q1ys,
        0
      )
    ));

    SYNC((
      GatherVertexCoordinates<<<nbBlock, trianglesPerBlock>>>(
        triangleVidsOnGpu,                                                              
        vertexCoordsOnGpu,
        q2xs,
        q2ys,
        1
      )
    ));

    SYNC((
      GatherVertexCoordinates<<<nbBlock, trianglesPerBlock>>>(
        triangleVidsOnGpu,
        vertexCoordsOnGpu,
        q3xs,
        q3ys,
        2
      )
    ));

    device_vector<float> coeffsOnGpu(9 * nt);
    size_t shmem = 9 * trianglesPerBlock * sizeof(float);
    SYNC((ComputeElementsCoeffs<<<nbBlock, trianglesPerBlock, shmem>>>(q1xs, q1ys, q2xs, q2ys, q3xs, q3ys, coeffsOnGpu)));

    device_vector<int> colorsOnGpu(colors, &colors[nt]);
    device_vector<int> ttOnGpu(tt, &tt[9 * nt]);
    device_vector<float> result(rowptr[nv]);
    fill(result.begin(), result.end(), 0.);
    device_vector<int>::iterator maxIter = max_element(colorsOnGpu.begin(), colorsOnGpu.end());
    int nbColor = 1 + *maxIter;
    for (int color = 0; color < nbColor; ++color)
    {
      SYNC((
        Assemble<<<nbBlock, trianglesPerBlock>>>( 
          coeffsOnGpu,
          colorsOnGpu,
          ttOnGpu,
          result,
          color
        )
      ));
    }

    copy(coeffsOnGpu.begin(), coeffsOnGpu.end(), coeffs);  
  }
}